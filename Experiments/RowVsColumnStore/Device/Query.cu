#include "hip/hip_runtime.h"
#include <Device/Query.cuh>
#include <stdio.h>

#define REQUIRE_SUCCESS(Expression) 																				\
{																													\
	const hipError_t error = Expression;																			\
	if (error != hipSuccess)																						\
	{																												\
		fprintf(stderr, "CUDA ERROR: %s:%d, code: %d, reason: %s\n",												\
				__FILE__, __LINE__, error, hipGetErrorString(error));												\
		return false;																								\
	}																												\
}

extern "C"
{

	const int N = 16;
	const int blocksize = 16;

	__global__
	void hello(char *a, int *b)
	{
		a[threadIdx.x] += b[threadIdx.x];
	}

	bool CopyDataToDevice(void **DestinationDevice, const void *SourceHost, size_t NumberOfBytes)
	{
		REQUIRE_SUCCESS(hipMalloc(DestinationDevice, NumberOfBytes));
		REQUIRE_SUCCESS(hipMemcpy(DestinationDevice, SourceHost, NumberOfBytes, hipMemcpyHostToDevice));
		return true;
	}

	bool CopyDataFromDevice(void **DestinationHost, const void *SourceDevice, size_t NumberOfBytes)
	{
		REQUIRE_SUCCESS(hipMemcpy(DestinationHost, SourceDevice, NumberOfBytes, hipMemcpyDeviceToDevice));
		return true;
	}

	bool FreeDataInDevice(void *DestinationDevice)
	{
		REQUIRE_SUCCESS(hipFree(DestinationDevice));
		return true;
	}


	void do_cuda_stuff()
	{
		char a[N] = "Hello \0\0\0\0\0\0";
		int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

		char *ad;
		int *bd;
		const int csize = N*sizeof(char);
		const int isize = N*sizeof(int);

		printf("%s", a);

		hipMalloc( (void**)&ad, csize );
		hipMalloc( (void**)&bd, isize );
		hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
		hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

		dim3 dimBlock( blocksize, 1 );
		dim3 dimGrid( 1, 1 );
		hello<<<dimGrid, dimBlock>>>(ad, bd);
		hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
		hipFree( ad );
		hipFree( bd );

		printf("%s\n", a);
	}

}
