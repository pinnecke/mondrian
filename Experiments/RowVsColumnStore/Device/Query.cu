#include "hip/hip_runtime.h"
#include <Device/Query.cuh>
#include "Kernels/ParallelReduction.cu"

#define REQUIRED(Expression, RetVal)																				\
{																													\
	const hipError_t error = Expression;																			\
	if (error != hipSuccess)																						\
	{																												\
		fprintf(stderr, "CUDA ERROR: %s:%d, code: %d\n\treason: %s\n\t%s\t",										\
				__FILE__, __LINE__, error, hipGetErrorString(error),#Expression);									\
		return RetVal;																								\
	}																												\
}

extern "C"
{
	DEVICE_MEM_HANDLE deviceCopyFromHostToDevice(const HOST_MEM_POINTER Source, size_t NumberOfBytes)
	{
		DEVICE_MEM_HANDLE deviceMemoryHandle;
		REQUIRED(hipMalloc(&deviceMemoryHandle, NumberOfBytes), NULL);
		REQUIRED(hipMemcpy(deviceMemoryHandle, Source, NumberOfBytes, hipMemcpyHostToDevice), NULL);
		hipDeviceSynchronize();
		return deviceMemoryHandle;
	}

	bool deviceCopyFromDeviceToHost(HOST_MEM_POINTER Destination, DEVICE_MEM_HANDLE Source, size_t NumberOfBytes)
	{
		REQUIRED(hipMemcpy(Destination, Source, NumberOfBytes, hipMemcpyDeviceToHost), false);
		return true;
	}


	DEVICE_MEM_HANDLE deviceQueryEvaluateSum(DEVICE_MEM_HANDLE PriceColumn, size_t NumberOfItems, bool MultipleThreads)
	{
		DEVICE_MEM_HANDLE out;

		int threads = MultipleThreads ? 512 : 1024;
		int blocks = MultipleThreads ? std::min((int)(NumberOfItems + threads - 1) / threads, 1024) : 1;

		REQUIRED(hipMalloc(&out, sizeof(size_t)*1024), 0);

		deviceReduceKernel<<<blocks, threads>>>((size_t*) PriceColumn, (size_t*) out, NumberOfItems);
	    deviceReduceKernel<<<1, 1024>>>((size_t*) out, (size_t*) out, blocks);
	    hipDeviceSynchronize();

	    return out;
	}

	size_t deviceQueryFetchSumValue(DEVICE_MEM_HANDLE deviceResultHandle)
	{
		size_t sum;
		REQUIRED(hipMemcpy(&sum,deviceResultHandle,sizeof(size_t),hipMemcpyDeviceToHost), 0);
		return sum;
	}

	void deviceCleanUp(DEVICE_MEM_HANDLE DevicePriceColumnHandle, DEVICE_MEM_HANDLE deviceResultHandle)
	{
		hipFree(DevicePriceColumnHandle);
		hipFree(deviceResultHandle);
		hipDeviceReset();
	}

}
