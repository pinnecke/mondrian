#include "hip/hip_runtime.h"
#include <Device/Query.cuh>
#include "Kernels/ParallelReduction.cu"

#define REQUIRED(Expression, RetVal)																				\
{																													\
	const hipError_t error = Expression;																			\
	if (error != hipSuccess)																						\
	{																												\
		fprintf(stderr, "CUDA ERROR: %s:%d, code: %d\n\treason: %s\n\t%s\t",										\
				__FILE__, __LINE__, error, hipGetErrorString(error),#Expression);									\
		return RetVal;																								\
	}																												\
}

extern "C"
{
	DEVICE_MEM_HANDLE deviceCopyFromHostToDevice(const HOST_MEM_POINTER Source, size_t NumberOfBytes)
	{
		DEVICE_MEM_HANDLE deviceMemoryHandle;
		REQUIRED(hipMalloc(&deviceMemoryHandle, NumberOfBytes), NULL);
		REQUIRED(hipMemcpy(deviceMemoryHandle, Source, NumberOfBytes, hipMemcpyHostToDevice), NULL);
		hipDeviceSynchronize();
		return deviceMemoryHandle;
	}

	bool deviceCopyFromDeviceToHost(HOST_MEM_POINTER Destination, DEVICE_MEM_HANDLE Source, size_t NumberOfBytes)
	{
		REQUIRED(hipMemcpy(Destination, Source, NumberOfBytes, hipMemcpyDeviceToHost), false);
		hipDeviceSynchronize();
		return true;
	}


	DEVICE_MEM_HANDLE deviceQueryEvaluateSum(DEVICE_MEM_HANDLE PriceColumn, size_t NumberOfItems, bool MultipleThreads)
	{
		DEVICE_MEM_HANDLE out;

		int threads = MultipleThreads ? 512 : 1024;
		int blocks = MultipleThreads ? std::min((int)(NumberOfItems + threads - 1) / threads, 1024) : 1;

		REQUIRED(hipMalloc(&out, sizeof(size_t)*1024), 0);

		deviceReduceKernel<<<blocks, threads>>>((size_t*) PriceColumn, (size_t*) out, NumberOfItems);
	    deviceReduceKernel<<<1, 1024>>>((size_t*) out, (size_t*) out, blocks);
	    hipDeviceSynchronize();

	    return out;
	}

	size_t deviceQueryFetchSumValue(DEVICE_MEM_HANDLE deviceResultHandle)
	{
		size_t sum;
		REQUIRED(hipMemcpy(&sum,deviceResultHandle,sizeof(size_t),hipMemcpyDeviceToHost), 0);
		return sum;
	}

	DEVICE_MEM_HANDLE *deviceQueryEvaluateManySum(DEVICE_MEM_HANDLE *PriceColumn, size_t NumberOfColumns, size_t NumberOfItems, bool MultipleThreads)
	{
		DEVICE_MEM_HANDLE *out = (DEVICE_MEM_HANDLE *) malloc(NumberOfColumns * sizeof(DEVICE_MEM_HANDLE));

		int threads = MultipleThreads ? 512 : 1024;
		int blocks = MultipleThreads ? std::min((int)(NumberOfItems + threads - 1) / threads, 1024) : 1;

		for (size_t i = 0; i < COLUMN_NUMBER_TO_COPY; ++i) {
			REQUIRED(hipMalloc(&out[i], sizeof(size_t)*1024), 0);
		}

		for (size_t i = 0; i < COLUMN_NUMBER_TO_COPY; ++i) {
			deviceReduceKernel<<<blocks, threads>>>((size_t*) PriceColumn[i], (size_t*) out[i], NumberOfItems); // TODO
			deviceReduceKernel<<<1, 1024>>>((size_t*) out[i], (size_t*) out[i], blocks); // TODO
		}

		hipDeviceSynchronize();

		return out;
	}

	size_t *deviceQueryFetchManySumValues(DEVICE_MEM_HANDLE *deviceResultHandle, size_t NumberOfColumns)
	{
		size_t *sum = (size_t *) malloc (sizeof(size_t) * NumberOfColumns);

		for (size_t i = 0; i < COLUMN_NUMBER_TO_COPY; ++i) {
			REQUIRED(hipMemcpy(&sum[i],deviceResultHandle[i],sizeof(size_t),hipMemcpyDeviceToHost), 0);
		}
		hipDeviceSynchronize();

		return sum;
	}

	void deviceCleanUp(DEVICE_MEM_HANDLE *DevHandle, size_t NumDevHandle)
	{
		for (size_t i = 0; i < NumDevHandle; i++) {
			REQUIRED(hipFree(DevHandle[i]), );
		}
		hipDeviceReset();
	}

}
